#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "image_processing.h"

__global__ void gaussianBlurKernel(const unsigned char* input, unsigned char* output, int width, int height, int channels, const float* filter, int filterWidth) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int pixelIndex = (row * width + col) * channels;
        float newValue = 0.0f;

        for (int filterRow = 0; filterRow < filterWidth; ++filterRow) {
            for (int filterCol = 0; filterCol < filterWidth; ++filterCol) {
                int imageRow = row + filterRow - filterWidth / 2;
                int imageCol = col + filterCol - filterWidth / 2;

                if (imageRow >= 0 && imageRow < height && imageCol >= 0 && imageCol < width) {
                    int imageIndex = (imageRow * width + imageCol) * channels;
                    newValue += input[imageIndex] * filter[filterRow * filterWidth + filterCol];
                }
            }
        }
        output[pixelIndex] = static_cast<unsigned char>(newValue);
    }
}

void applyGaussianBlur(const unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int imageSize = width * height * channels;
    unsigned char* d_input;
    unsigned char* d_output;
    float h_filter[] = {1/16.0f, 2/16.0f, 1/16.0f, 2/16.0f, 4/16.0f, 2/16.0f, 1/16.0f, 2/16.0f, 1/16.0f};
    float* d_filter;

    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, imageSize);
    hipMalloc(&d_filter, sizeof(h_filter));

    hipMemcpy(d_input, input, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, sizeof(h_filter), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    gaussianBlurKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height, channels, d_filter, 3);

    hipMemcpy(output, d_output, imageSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_filter);
}
